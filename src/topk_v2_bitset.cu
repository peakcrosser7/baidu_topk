#include "hip/hip_runtime.h"

#include "topk.h"
#include "thread_pool.h"

#include <hipcub/hipcub.hpp>
#include <cuda_fp16.hpp>
#include <chrono>
#include <numeric>
#include <cuda_pipeline.h>

#include <emmintrin.h>
#include <mmintrin.h>

#include "fast_topk.cuh"

typedef uint4 group_t;

constexpr static const int TOPK = 100;
constexpr static const int N_THREADS_IN_ONE_BLOCK = 512;
constexpr static const int MAX_DOC_SIZE = 128;

constexpr static const int max_batch = 4;
// constexpr static const int max_id = 50000;
constexpr static const int query_mask_size = 1568;  // 1568 * 32 > 50000
constexpr static const int default_sort_storage = 64 * 1024 * 1024;
constexpr static const int num_threads = 8;

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
        const uint16_t *docs, 
        const int *doc_lens,
        const size_t n_docs, 
        uint32_t *query,
        const uint16_t max_query_token,
        const int query_len,
        float *scores) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int threadid = threadIdx.x;

    __shared__ uint32_t query_mask[query_mask_size];

    #pragma unroll
    for (int l = threadid; l < query_mask_size; l += N_THREADS_IN_ONE_BLOCK) {
        query_mask[l] = __ldg(query + l);
    }
    __syncthreads();

    if (tid >= n_docs) {
        return;
    }

    int doc_id = tid;
    int doc_len = doc_lens[doc_id];
    int loop = (doc_len + 7) / 8;

    uint16_t tmp_score = 0;

    for (int i = 0; i < loop; ++i) {
        group_t loaded = ((group_t*)docs)[i * n_docs + doc_id];
        uint16_t* token = (uint16_t*)(&loaded);

        #pragma unroll
        for (auto j = 0; j < 8; ++j) {
            uint16_t tindex = token[j] >> 5;
            uint16_t tpos = token[j] & 31;

            tmp_score += (query_mask[tindex] >> tpos) & 0x01;
            // tmp_score += __popc(query_mask[tindex] & tmask);
        }

        if (token[7] >= max_query_token) {
            break;
        }
    }
    scores[doc_id] = 1.f * tmp_score / max(query_len, doc_len);
}

#define MYTIME

#ifdef MYTIME
struct Timer {
    const char* m_name;
    std::chrono::high_resolution_clock::time_point m_start;
    std::chrono::high_resolution_clock::time_point m_stop;

    Timer(const char* name) {
        m_name = name;
        m_start = std::chrono::high_resolution_clock::now();
    }

    void stop(const char* name = nullptr) {
        // CHECK_CUDA(hipDeviceSynchronize());
        m_stop = std::chrono::high_resolution_clock::now();
        double cur_time = std::chrono::duration<double, std::milli>(m_stop-m_start).count();
        printf("==== %s: %.3fms\n", m_name, cur_time);

        m_name = name;
        m_start = std::chrono::high_resolution_clock::now();
    }
};
#else
struct Timer {
    Timer(const char* name) {}
    void stop(const char* name = nullptr) {}
};
#endif

struct MemsetTask : public Task {
    MemsetTask(uint16_t* ptr, size_t size)
        : m_ptr(ptr), m_size(size) 
    {}

    void run() override {
        memset(m_ptr, 0, m_size);
    }

    uint16_t* m_ptr = nullptr;
    size_t m_size = 0;
};

struct HostCopyTask : public Task {
    HostCopyTask(int start, int end, std::vector<int>& h_doc_lens_vec, uint16_t* h_docs, std::vector<std::vector<uint16_t>> & docs)
        : m_start(start), m_end(end), m_h_doc_lens(h_doc_lens_vec), m_h_docs(h_docs), m_docs(docs)
    {}

    void run() override {
        auto group_sz = sizeof(group_t) / sizeof(uint16_t);
        auto n_docs = m_docs.size();
        auto layer_0_stride = n_docs * group_sz;
        auto layer_1_stride = group_sz;
        for (int i = m_start; i < m_end; i++) {
            auto layer_1_offset = i;
            auto layer_1_offset_val = layer_1_offset * layer_1_stride;
            for (int j = 0; j < m_docs[i].size(); j++) {
                auto layer_0_offset = j / group_sz;
                auto layer_2_offset = j % group_sz;
                auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset_val + layer_2_offset;
                m_h_docs[final_offset] = m_docs[i][j];
            }
            m_h_doc_lens[i] = m_docs[i].size();
        }
    }

    int m_start = 0;
    int m_end = 0;
    std::vector<int>& m_h_doc_lens;
    uint16_t* m_h_docs;
    std::vector<std::vector<uint16_t>> & m_docs;
};

struct TopkTask : public Task {

    TopkTask(int start, int end, std::vector<std::vector<uint16_t>> &querys,
            uint16_t *d_docs, int *d_doc_lens, int n_docs, std::vector<std::vector<int>> &indices)
        : m_start(start), m_end(end), m_querys(querys), m_d_docs(d_docs), m_d_doc_lens(d_doc_lens),
          m_n_docs(n_docs), m_indices(indices) {}

    void run() override {
        if (m_start >= m_end) {
            return;
        }

        hipDeviceProp_t device_props;
        hipGetDeviceProperties(&device_props, 0);
        hipSetDevice(0);

        hipStream_t stream;
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        uint32_t* d_query;
        hipMallocAsync(&d_query, sizeof(uint32_t) * query_mask_size, stream);
        float* d_scores;
        hipMallocAsync(&d_scores, sizeof(float) * m_n_docs, stream);

        std::vector<int> indices(m_n_docs);
        for (int i = 0; i < m_n_docs; ++i) {
            indices[i] = i;
        }

        // int* s_indices;
        // hipHostMalloc(&s_indices, n_docs * sizeof(int));
        // float* scores;
        // hipHostMalloc(&scores, n_docs * sizeof(float));
        std::vector<int> s_indices(m_n_docs);
        std::vector<float> scores(m_n_docs);

        for (int i = m_start; i < m_end; ++i) {
            auto& query = m_querys[i];
            //init indices
            memcpy(s_indices.data(), indices.data(), indices.size() * sizeof(int));

            const size_t query_len = query.size();
            std::vector<uint32_t> query_mask(query_mask_size, 0u);
            for (auto& q : query) {
                int index = q / 32;
                int postion = q % 32;
                query_mask[index] |= ((1u) << postion);    
            }
            hipMemcpyAsync(d_query, query_mask.data(), sizeof(uint32_t) * query_mask_size, hipMemcpyHostToDevice, stream);

            // launch kernel
            int block = N_THREADS_IN_ONE_BLOCK;
            int grid = (m_n_docs + block - 1) / block;
            uint16_t max_query_token = query.back();

            docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0, stream>>>(m_d_docs,
                m_d_doc_lens, m_n_docs, d_query, max_query_token, query_len, d_scores);

            // hipDeviceSynchronize();
            hipMemcpyAsync(scores.data(), d_scores, sizeof(float) * m_n_docs, hipMemcpyDeviceToHost, stream);
            hipStreamSynchronize(stream);

            // sort scores
            std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK, s_indices.end(),
                            [&scores](const int& a, const int& b) {
                                if (scores[a] != scores[b]) {
                                    return scores[a] > scores[b];  // 按照分数降序排序
                                }
                                return a < b;  // 如果分数相同，按索引从小到大排序
                        });
            std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
            m_indices[i] = std::move(s_ans);
            // hipFree(d_query);
        }

        hipFreeAsync(d_query, stream);
        hipFreeAsync(d_scores, stream);
        hipStreamDestroy(stream);
    }

    int m_start = 0;
    int m_end = 0;
    std::vector<std::vector<uint16_t>> & m_querys;
    uint16_t* m_d_docs = nullptr;
    int* m_d_doc_lens = nullptr;
    int m_n_docs = 0;
    std::vector<std::vector<int>> & m_indices;
};

void doc_query_scoring_gpu_function(std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs,
    std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices //shape [querys.size(), TOPK]
    ) {

    auto n_docs = docs.size();
    std::vector<float> scores(n_docs);
    std::vector<int> s_indices(n_docs);

    float *d_scores = nullptr;
    uint16_t *d_docs = nullptr;
    uint32_t *d_query = nullptr;
    int *d_doc_lens = nullptr;

    ThreadPool pool;
    int num_threads = min(8, static_cast<int>(n_docs));
    pool.set_num_threads(num_threads);

Timer t("pre_process");

    // copy to device
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    hipMalloc(&d_scores, sizeof(float) * n_docs);
    hipMalloc(&d_doc_lens, sizeof(int) * n_docs);

    uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
#if 0
    memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
#else
    std::vector<Task*> tasks(num_threads, nullptr);
    size_t n_docs_per_threads = (n_docs + num_threads - 1) / num_threads;
    int offset = 0;
    for (int i = 0; i < num_threads; ++i) {
        int size = min(n_docs_per_threads, n_docs - offset) * sizeof(uint16_t) * MAX_DOC_SIZE;
        tasks[i] = new MemsetTask(h_docs + MAX_DOC_SIZE * offset, size);
        offset += n_docs_per_threads;
    }
    pool.run_task(tasks);
    pool.wait();
#endif

    std::vector<int> h_doc_lens_vec(n_docs);
#if 1
    std::vector<Task*> host_copy_tasks(num_threads, nullptr);
    // size_t n_docs_per_threads = (n_docs + num_threads - 1) / num_threads;
    offset = 0;
    for (int i = 0; i < num_threads; ++i) {
        int size = min(n_docs_per_threads, n_docs - offset);
        int end = offset + size;
        host_copy_tasks[i] = new HostCopyTask(offset, end, h_doc_lens_vec, h_docs, docs);
        offset += n_docs_per_threads;
    }
    pool.run_task(host_copy_tasks);
    pool.wait();
#else
    for (int i = 0; i < docs.size(); i++) {
        for (int j = 0; j < docs[i].size(); j++) {
            auto group_sz = sizeof(group_t) / sizeof(uint16_t);
            auto layer_0_stride = n_docs * group_sz;
            auto layer_0_offset = j / group_sz;
            auto layer_1_offset = i;
            auto layer_1_stride = group_sz;
            auto layer_2_offset = j % group_sz;
            auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
            h_docs[final_offset] = docs[i][j];
        }
        h_doc_lens_vec[i] = docs[i].size();
    }
#endif

    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    hipMemcpy(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs, hipMemcpyHostToDevice);

    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);

    hipSetDevice(0);

t.stop("topk");

#if 0
    for(auto& query : querys) {
        //init indices
        for (int i = 0; i < n_docs; ++i) {
            s_indices[i] = i;
        }

        const size_t query_len = query.size();
        hipMalloc(&d_query, sizeof(uint32_t) * query_mask_size);
        std::vector<uint32_t> query_mask(query_mask_size, 0u);
        for (auto& q : query) {
            int index = q / 32;
            int postion = q % 32;
            query_mask[index] |= ((1u) << postion);    
        }
        hipMemcpy(d_query, query_mask.data(), sizeof(uint32_t) * query_mask_size, hipMemcpyHostToDevice);

        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;
        uint16_t max_query_token = query.back();
        docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block>>>(d_docs,
            d_doc_lens, n_docs, d_query, max_query_token, query_len, d_scores);

        // hipDeviceSynchronize();
        hipMemcpy(scores.data(), d_scores, sizeof(float) * n_docs, hipMemcpyDeviceToHost);

        // sort scores
        std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK, s_indices.end(),
                        [&scores](const int& a, const int& b) {
                            if (scores[a] != scores[b]) {
                                return scores[a] > scores[b];  // 按照分数降序排序
                            }
                            return a < b;  // 如果分数相同，按索引从小到大排序
                    });
        std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
        indices.push_back(s_ans);

        hipFree(d_query);
    }
#else
    indices.resize(querys.size());
    std::vector<Task*> topk_tasks(num_threads, nullptr);
    int num_querys = querys.size();
    int n_query_per_threads = (num_querys + num_threads - 1) / num_threads;
    int start = 0;
    for (int i = 0; i < num_threads; ++i) {
        int size = min(n_query_per_threads, num_querys - start);
        int end = start + size;
        topk_tasks[i] = new TopkTask(start, end, querys, d_docs, d_doc_lens, n_docs, indices);
        start = end;
    }
    pool.run_task(topk_tasks);
    pool.wait();
#endif

t.stop();

    // deallocation
    hipFree(d_docs);
    //hipFree(d_query);
    hipFree(d_scores);
    hipFree(d_doc_lens);
    free(h_docs);
}