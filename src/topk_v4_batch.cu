#include "hip/hip_runtime.h"

#include "topk.h"
#include "thread_pool.h"

#include <hipcub/hipcub.hpp>
#include <cuda_fp16.hpp>
#include <chrono>
#include <numeric>
#include <cuda_pipeline.h>

#include <emmintrin.h>
#include <mmintrin.h>

#include "fast_topk.cuh"

typedef uint4 group_t;
constexpr static const int TOPK = 100;
constexpr static const int N_THREADS_IN_ONE_BLOCK = 512;
constexpr static const int MAX_DOC_SIZE = 128;

constexpr static const int max_batch = 4;
constexpr static const int max_id = 50000;
constexpr static const int query_mask_size = 1568;  // 1568 * 32 > 50000
constexpr static const int default_sort_storage = 64 * 1024 * 1024;
constexpr static const int num_threads = 8;

template <int N>
struct PackData {};

template <>
struct PackData<1> {
    using dtype = uint32_t;
};

template <>
struct PackData<2> {
    using dtype = uint2;
};

template <>
struct PackData<3> {
    using dtype = uint3;
};

template <>
struct PackData<4> {
    using dtype = uint4;
};

__device__ __forceinline__
uint32_t getBitfield(uint32_t val, int pos, int len) {
    uint32_t ret;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(val), "r"(pos), "r"(len));
    return ret;
}

__device__ __forceinline__
uint64_t getBitfield64(uint64_t val, int pos, int len) {
    uint64_t ret;
    asm("bfe.u64 %0, %1, %2, %3;" : "=l"(ret) : "l"(val), "r"(pos), "r"(len));
    return ret;
}

template<int N=4>
#if __CUDA_ARCH__ == 860
__launch_bounds__(N_THREADS_IN_ONE_BLOCK, 3)
#elif __CUDA_ARCH__ == 800
__launch_bounds__(N_THREADS_IN_ONE_BLOCK, 4)
#endif
void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
        const uint16_t* docs,
        const uint16_t* doc_lens,
        const size_t n_docs, 
        const uint32_t* query,
        const uint16_t* query_len,
        const uint16_t max_query_token,
        int16_t *scores) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    __shared__ uint32_t query_mask[N * query_mask_size];
    // __shared__ group_t doc_cache[N_THREADS_IN_ONE_BLOCK];

    int threadid = threadIdx.x;

    #pragma unroll
    for (int l = threadid; l < N * query_mask_size; l += N_THREADS_IN_ONE_BLOCK) {
        query_mask[l] = __ldg(query + l);
    }
    __syncthreads();

    for (int doc_id = tid; doc_id < n_docs; doc_id += stride) {
        int doc_len = doc_lens[doc_id];
        int loop = (doc_len + 7) / 8;

        int tmp_score[N] = {0};

        for (int i = 0; i < loop; ++i) {

            group_t loaded = ((group_t*)docs)[i * n_docs + doc_id];
            uint16_t* token = (uint16_t*)(&loaded);

            #pragma unroll
            for (auto j = 0; j < 8; ++j) {
                uint16_t tindex = token[j] >> 5;
                uint16_t tpos = token[j] & 31;

                #pragma unroll
                for (auto k = 0; k < N; ++k) {
                    tmp_score[k] += (query_mask[k * query_mask_size + tindex] >> tpos) & 0x01;
                }
            }

            if (token[7] >= max_query_token) {
                break;
            }
        }

        for (auto i = 0; i < N; ++i) {
            scores[i * n_docs + doc_id] = static_cast<int16_t>(
                1.f * 128 * 128 * tmp_score[i] / max(query_len[i], doc_len));
        }
    }
}

void search_topk(
        int n_docs_pad,
        uint16_t* d_docs,
        uint16_t* d_doc_lens,
        int16_t* d_scores,
        Pair* d_topk,
        uint32_t* d_query,
        uint16_t* d_query_len,
        void* d_temp_storage,
        Pair* h_topk,
        uint32_t* h_query,
        uint16_t* h_query_len,
        std::vector<std::vector<uint16_t>> &querys,
        std::vector<std::vector<int>> &indices,
        int start,
        int batch,
        hipStream_t stream) {

    int cur_batch = batch;
    memset(h_query, 0, sizeof(uint32_t) * cur_batch * query_mask_size);
    uint16_t max_query_token = 0;
    for (int j = 0; j < cur_batch; ++j) {
        auto& query = querys[start + j];
        h_query_len[j] = query.size();
        for (auto& q : query) {
            uint16_t index = q >> 5;
            uint16_t postion = q & 31;
            h_query[j * query_mask_size + index] |= ((1u) << postion);
            // h_query[cur_batch * index + j] |= ((1u) << postion);
        }
        max_query_token = std::max(max_query_token, query.back());
    }

    CHECK_CUDA(hipMemcpyAsync(d_query, h_query,
            cur_batch * query_mask_size * sizeof(uint32_t),
            hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_query_len, h_query_len,
            cur_batch * sizeof(uint16_t),
            hipMemcpyHostToDevice, stream));

    int block = N_THREADS_IN_ONE_BLOCK;
    int grid = n_docs_pad / 4096;

    if (cur_batch == 4) {
        docQueryScoringCoalescedMemoryAccessSampleKernel<4><<<grid, block, 0, stream>>>(
                d_docs, d_doc_lens, n_docs_pad, d_query, d_query_len, max_query_token, d_scores);
        CHECK_CUDA(hipGetLastError());
    } else if (cur_batch == 3) {
        docQueryScoringCoalescedMemoryAccessSampleKernel<3><<<grid, block, 0, stream>>>(
                d_docs, d_doc_lens, n_docs_pad, d_query, d_query_len, max_query_token, d_scores);
        CHECK_CUDA(hipGetLastError());
    } else if (cur_batch == 2) {
        docQueryScoringCoalescedMemoryAccessSampleKernel<2><<<grid, block, 0, stream>>>(
                d_docs, d_doc_lens, n_docs_pad, d_query, d_query_len, max_query_token, d_scores);
        CHECK_CUDA(hipGetLastError());
    } else if (cur_batch == 1) {
        docQueryScoringCoalescedMemoryAccessSampleKernel<1><<<grid, block, 0, stream>>>(
                d_docs, d_doc_lens, n_docs_pad, d_query, d_query_len, max_query_token, d_scores);
        CHECK_CUDA(hipGetLastError());
    }

    launch_gather_topk_kernel(
            d_scores, d_topk, (int8_t*)d_temp_storage, TOPK, cur_batch, n_docs_pad, stream);
    CHECK_CUDA(hipMemcpyAsync(h_topk, d_topk, cur_batch * TOPK * sizeof(Pair), hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));
    for (int j = 0; j < cur_batch; ++j) {
        Pair* cur_topk = h_topk + j * TOPK;
        std::vector<int> s_ans(TOPK);
        std::sort(cur_topk, cur_topk + TOPK,
                [](const Pair& a, const Pair& b) {
                    if (a.score != b.score) {
                        return a.score > b.score;
                    }
                    return a.index < b.index;
                });
        for (int k = 0; k < TOPK; ++k) {
            s_ans[k] = cur_topk[k].index;
        }
        indices[start + j] = std::move(s_ans);
    }
}

#define MYTIME

#ifdef MYTIME
struct Timer {
    const char* m_name;
    std::chrono::high_resolution_clock::time_point m_start;
    std::chrono::high_resolution_clock::time_point m_stop;

    Timer(const char* name) {
        m_name = name;
        m_start = std::chrono::high_resolution_clock::now();
    }

    void stop(const char* name = nullptr) {
        // CHECK_CUDA(hipDeviceSynchronize());
        m_stop = std::chrono::high_resolution_clock::now();
        double cur_time = std::chrono::duration<double, std::milli>(m_stop-m_start).count();
        printf("==== %s: %.3fms\n", m_name, cur_time);

        m_name = name;
        m_start = std::chrono::high_resolution_clock::now();
    }
};
#else
struct Timer {
    Timer(const char* name) {}
    void stop(const char* name = nullptr) {}
};
#endif

struct Context {
    Context() = default;

    void init(int n_docs, int num_threads) {
Timer t("init");
        pool.set_num_threads(num_threads);
        
        CHECK_CUDA(hipSetDevice(0));
        CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

        thread_contexts.resize(num_threads);
        for (int i = 0; i < num_threads; ++i) {
            CHECK_CUDA(hipStreamCreateWithFlags(&thread_contexts[i].stream, hipStreamNonBlocking));
        }

t.stop("cuda_malloc_device");
        // 计算好需要分配的显存大小
        size_t bytes = 0u;
        bytes += align_bytes(sizeof(uint16_t) * 128 * n_docs);                      // d_docs
        bytes += align_bytes(sizeof(uint16_t) * n_docs);                            // d_doc_lens
        for (int i = 0; i < num_threads; ++i) {
            bytes += align_bytes(sizeof(half) * max_batch * n_docs);                // d_scores
            bytes += align_bytes(sizeof(Pair) * max_batch * TOPK);                  // d_topk
            bytes += align_bytes(sizeof(uint32_t) * max_batch * query_mask_size);   // d_query
            bytes += align_bytes(sizeof(uint32_t) * max_batch);                     // d_query_len
            bytes += align_bytes(default_sort_storage);                             // d_temp_storage
        }
        CHECK_CUDA(hipMalloc(&d_mem, bytes));

t.stop("cuda_malloc_host");
        auto ret = posix_memalign((void**)(&h_mem), 256, sizeof(uint16_t) * 128 * n_docs);
        (void)(ret);

        int8_t* h_mem_pool = h_mem;
        int8_t* d_mem_pool = d_mem;
        // 初始化指针
        h_docs = reinterpret_cast<uint16_t*>(h_mem_pool);
        h_mem_pool += align_bytes(sizeof(uint16_t) * 128 * n_docs);

        d_docs = reinterpret_cast<uint16_t*>(d_mem_pool);
        d_mem_pool += align_bytes(sizeof(uint16_t) * 128 * n_docs);
        d_doc_lens = reinterpret_cast<uint16_t*>(d_mem_pool);
        d_mem_pool += align_bytes(sizeof(uint16_t) * n_docs);

        for (int i = 0; i < num_threads; ++i) {
            ThreadContext& ctx = thread_contexts[i];
            ctx.d_scores = reinterpret_cast<int16_t*>(d_mem_pool);
            d_mem_pool += align_bytes(sizeof(int16_t) * max_batch * n_docs);
            ctx.d_topk = reinterpret_cast<Pair*>(d_mem_pool);
            d_mem_pool += align_bytes(sizeof(Pair) * max_batch * TOPK);
            ctx.d_query = reinterpret_cast<uint32_t*>(d_mem_pool);
            d_mem_pool += align_bytes(sizeof(uint32_t) * max_batch * query_mask_size);
            ctx.d_query_len = reinterpret_cast<uint16_t*>(d_mem_pool);
            d_mem_pool += align_bytes(sizeof(uint16_t) * max_batch);
            ctx.d_temp_storage = reinterpret_cast<void*>(d_mem_pool);
            d_mem_pool += align_bytes(default_sort_storage);
        }
t.stop("thread_pool");
        pool.wait();
t.stop();
    }

    void init_pinned(int n_docs, int num_threads) {
Timer t("cuda_malloc_pinned");
        // 计算需要分配的 pinned 内存大小
        // 由于 hipHostMalloc 分配大块内存时特别耗时, 所以 h_docs 空间使用 malloc 分配
        size_t bytes = 0u;
        
        for (int i = 0; i < num_threads; ++i) {
            bytes += align_bytes(sizeof(Pair) * max_batch * TOPK);                  // h_topk
            bytes += align_bytes(sizeof(uint32_t) * max_batch * query_mask_size);   // h_query
            bytes += align_bytes(sizeof(uint32_t) * max_batch);                     // h_query_len
            // bytes += align_bytes(sizeof(uint16_t) * 1 * n_docs);                    // h_scores
        }
        CHECK_CUDA(hipHostMalloc(&h_pinned_mem, bytes));

        int8_t* h_pinned_mem_pool = h_pinned_mem;

        for (int i = 0; i < num_threads; ++i) {
            ThreadContext& ctx = thread_contexts[i];

            ctx.h_topk = reinterpret_cast<Pair*>(h_pinned_mem_pool);
            h_pinned_mem_pool += align_bytes(sizeof(Pair) * max_batch * TOPK);
            ctx.h_query = reinterpret_cast<uint32_t*>(h_pinned_mem_pool);
            h_pinned_mem_pool += align_bytes(sizeof(uint32_t) * max_batch * query_mask_size);
            ctx.h_query_len = reinterpret_cast<uint16_t*>(h_pinned_mem_pool);
            h_pinned_mem_pool += align_bytes(sizeof(uint16_t) * max_batch);
            // ctx.h_scores = reinterpret_cast<int16_t*>(h_pinned_mem_pool);
            // h_pinned_mem_pool += align_bytes(sizeof(int16_t) * 1 * n_docs);
        }
t.stop();
    }

    void clear() {
        if (d_mem) {
            hipFree(d_mem);
        }
        if (h_mem) {
            // hipHostFree(h_mem);
            free(h_mem);
        }

        if (h_pinned_mem) {
            hipHostFree(h_pinned_mem);
        }

        hipStreamDestroy(stream);
        for (int i = 0; i < num_threads; ++i) {
            hipStreamDestroy(thread_contexts[i].stream);
        }
    }

    // init
    int8_t* d_mem = nullptr;
    int8_t* h_mem = nullptr;
    int8_t* h_pinned_mem = nullptr;
    hipStream_t stream;
    hipDeviceProp_t prop;
    ThreadPool pool;

    struct ThreadContext {
        hipStream_t stream;
        int16_t* d_scores = nullptr;        // [max_batch * n_docs]
        Pair* d_topk = nullptr;             // [max_batch * TOPK]
        uint32_t* d_query = nullptr;        // [max_batch * query_mask_size]
        uint16_t* d_query_len = nullptr;    // [max_batch]
        void* d_temp_storage = nullptr;     // [64 * 1024 * 1024]

        Pair* h_topk = nullptr;             // [max_batch * TOPK]
        uint32_t* h_query = nullptr;        // [max_batch * query_mask_size]
        uint16_t* h_query_len = nullptr;    // [max_batch]
        int16_t* h_scores = nullptr;        // [max_batch * n_docs]
    };

    // update
    uint16_t* h_docs = nullptr;             // [16, n_docs, 8]
    uint16_t* d_docs = nullptr;             // [16, n_docs, 8]
    uint16_t* d_doc_lens = nullptr;         // [n_docs]
    std::vector<ThreadContext> thread_contexts;
};

struct HostCopyTask : public Task {
    HostCopyTask(
            Context& ctx_,
            int id_,
            int world_,
            int start_,
            int end_,
            int n_docs_pad_,
            uint16_t* h_docs_,
            std::vector<std::vector<uint16_t>> & docs_)
        : ctx(ctx_),
          id(id_),
          world(world_),
          start(start_),
          end(end_),
          n_docs_pad(n_docs_pad_),
          h_docs(h_docs_),
          docs(docs_) {}

    void run() override {

Timer t("host_copy");
        auto group_sz = sizeof(group_t) / sizeof(uint16_t);
        auto layer_0_stride = n_docs_pad * group_sz;
        auto layer_1_stride = group_sz;
        int max_len = 0;
        for (int i = start; i < end; i++) {
            auto layer_1_offset = i;

            int doc_len = docs[i].size();
            max_len = std::max(doc_len, max_len);

            int n = doc_len / 8;
            int leftover = doc_len % 8;
            uint16_t * ptr = docs[i].data();

            int offset = layer_1_offset * layer_1_stride;
            for (int j = 0; j < n; ++j) {
                __m128i a = _mm_loadu_si128((__m128i*)(ptr));
                _mm_store_si128((__m128i*)(h_docs + offset), a);
                ptr += 8;
                offset += layer_0_stride;
            }
            if (leftover) {
                // 此处补 0 避免了对整个 h_docs 进行置 0 操作
                alignas(16) int16_t data[8] = {0, 0, 0, 0, 0, 0, 0, 0};
                for (int j = 0; j < leftover; ++j) {
                    data[j] = ptr[j];
                }
                __m128i a = _mm_load_si128((__m128i*)(data));
                _mm_store_si128((__m128i*)(h_docs + offset), a);
            }
        }
t.stop("device_copy");

        // [16, n_docs_pad, 8]
        uint16_t* d_docs = ctx.d_docs;
        hipStream_t stream = ctx.thread_contexts[id].stream;
        // 并不需要拷贝所有的 16 个 group 的数据, 可以只拷贝有效的数值, 在 kernel 中也只会读取有效的部分数据
        int loop = (max_len + 7) / 8;
        for (int i = 0; i < loop; ++i) {
            CHECK_CUDA(hipMemcpyAsync(d_docs + i * n_docs_pad * 8 + start * 8,
                    h_docs + i * n_docs_pad * 8 + start * 8,
                    (end - start) * 8 * sizeof(uint16_t),
                    hipMemcpyHostToDevice, stream));
        }
        CHECK_CUDA(hipStreamSynchronize(stream));
t.stop();
    }

    Context& ctx;
    int id = 0;
    int world = 0;
    int start = 0;
    int end = 0;
    int n_docs_pad = 0;
    uint16_t* h_docs;
    std::vector<std::vector<uint16_t>> & docs;
};

struct TopkTask : public Task {

    TopkTask(
            Context& ctx_,
            int id_,
            int world_,
            int start_,
            int end_,
            std::vector<std::vector<uint16_t>> &querys_,
            std::vector<int> &query_idx_,
            uint16_t *d_docs_,
            uint16_t *d_doc_lens_,
            int n_docs_pad_,
            std::vector<std::vector<int>> &indices_)
        : ctx(ctx_),
          id(id_),
          world(world_),
          start(start_),
          end(end_),
          querys(querys_),
          query_idx(query_idx_),
          d_docs(d_docs_),
          d_doc_lens(d_doc_lens_),
          n_docs_pad(n_docs_pad_),
          indices(indices_) {}

    void run() override {
        if (start >= end) {
            return;
        }

        Context::ThreadContext& tctx = ctx.thread_contexts[id];
        hipStream_t stream = tctx.stream;

        int16_t* d_scores = tctx.d_scores;
        Pair* d_topk = tctx.d_topk;
        uint32_t* d_query = tctx.d_query;
        uint16_t* d_query_len = tctx.d_query_len;
        void* d_temp_storage = tctx.d_temp_storage;

        Pair* h_topk = tctx.h_topk;
        uint32_t* h_query = tctx.h_query;
        uint16_t* h_query_len = tctx.h_query_len;

        int total_items = querys.size();
        for (int i = max_batch * id; i < total_items; i += max_batch * world) {
            int cur_batch = std::min<int>(total_items - i, max_batch);

            search_topk(n_docs_pad,
                        d_docs,
                        d_doc_lens,
                        d_scores,
                        d_topk,
                        d_query,
                        d_query_len,
                        d_temp_storage,
                        h_topk,
                        h_query,
                        h_query_len,
                        querys,
                        indices,
                        i,
                        cur_batch,
                        stream);
        }
    }

    Context& ctx;
    int id = 0;
    int world = 0;
    int start = 0;
    int end = 0;
    std::vector<std::vector<uint16_t>> & querys;
    std::vector<int> & query_idx;
    uint16_t* d_docs = nullptr;
    uint16_t* d_doc_lens = nullptr;
    int n_docs_pad = 0;
    std::vector<std::vector<int>> & indices;
};

void doc_query_scoring_gpu_function(std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs,
    std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices //shape [querys.size(), TOPK]
    ) {

Timer t("pre_malloc_host");
    size_t n_docs = docs.size();
    size_t n_docs_pad = (n_docs + 4095) / 4096 * 4096;

    // 分配资源，包括显存、内存、流、线程池等
    Context ctx;
    ctx.init(n_docs_pad, num_threads);

    uint16_t* h_docs = ctx.h_docs;

t.stop("pre_thread_pool");
    ThreadPool& pool = ctx.pool;

    // 使用多线程来将分散的 docs 拷贝到连续的 h_docs 内存中
    // 每个线程会将自己处理的内存拷贝到 d_docs 中
    // 这样可以避免主线程需要拷贝一大块空间到显存
    std::vector<Task*> tasks(num_threads, nullptr);
    size_t n_docs_per_threads = (n_docs_pad + num_threads - 1) / num_threads;
    int offset = 0;
    for (int i = 0; i < num_threads; ++i) {
        int size = min(n_docs_per_threads, n_docs_pad - offset);
        int end = offset + size;
        tasks[i] = new HostCopyTask(ctx, i, num_threads, offset, end, n_docs_pad, h_docs, docs);
        offset += n_docs_per_threads;
    }
    pool.run_task(tasks);

    ctx.init_pinned(n_docs_pad, num_threads);

    // 线程池在处理 d_docs 时, 主线程处理其他耗时的操作
t.stop("pre_init_cuda");

    std::vector<int> query_idx(querys.size());
    std::iota(query_idx.begin(), query_idx.end(), 0);
    std::sort(query_idx.begin(), query_idx.end(),
            [&querys](int a, int b) {
                return querys[a].back() < querys[b].back();
            });
t.stop("pre_malloc_device");
    hipStream_t stream = ctx.stream;
    uint16_t* d_docs = ctx.d_docs;
    uint16_t* d_doc_lens = ctx.d_doc_lens;
    CHECK_CUDA(hipMemcpyAsync(d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs,
            hipMemcpyHostToDevice, stream));
    if (n_docs != n_docs_pad) {
        CHECK_CUDA(hipMemsetAsync(d_doc_lens + n_docs, 0,
                (n_docs_pad - n_docs) * sizeof(uint16_t), stream));
    }
    CHECK_CUDA(hipStreamSynchronize(stream));
    indices.resize(querys.size());

t.stop("pre_memcpy_device");

    // 等待线程池完成 docs-> h_docs -> d_docs 的任务
    pool.wait();
t.stop("topk");
    // if (false) {
    if (true) {
        // 单线程处理 query
        Context::ThreadContext& tctx = ctx.thread_contexts[0];
        int16_t* d_scores = tctx.d_scores;
        Pair* d_topk = tctx.d_topk;
        uint32_t* d_query = tctx.d_query;
        uint16_t* d_query_len = tctx.d_query_len;
        void* d_temp_storage = tctx.d_temp_storage;

        Pair* h_topk = tctx.h_topk;
        uint32_t* h_query = tctx.h_query;
        uint16_t* h_query_len = tctx.h_query_len;

        for (int i = 0; i < querys.size(); i += max_batch) {
            int cur_batch = std::min<int>(querys.size() - i, max_batch);

// Timer tt("query");
            search_topk(n_docs_pad,
                        d_docs,
                        d_doc_lens,
                        d_scores,
                        d_topk,
                        d_query,
                        d_query_len,
                        d_temp_storage,
                        h_topk,
                        h_query,
                        h_query_len,
                        querys,
                        indices,
                        i,
                        cur_batch,
                        stream);
// tt.stop();
        }
    } else {
        // 多线程处理 query
        std::vector<Task*> topk_tasks(num_threads);
        int num_querys = querys.size();
        int n_query_per_threads = (num_querys + num_threads - 1) / num_threads;
        int start = 0;
        for (int i = 0; i < num_threads; ++i) {
            int size = min(n_query_per_threads, num_querys - start);
            int end = start + size;
            topk_tasks[i] = new TopkTask(
                    ctx, i, num_threads, start, end, querys, query_idx, d_docs, d_doc_lens, n_docs_pad, indices);
            start = end;
        }

        pool.run_task(topk_tasks);
        pool.wait();
    }
t.stop();
}